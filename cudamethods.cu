#include "hip/hip_runtime.h"

#include "cudamethods.hu"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdarg.h>

CudaQueueElement *first = NULL;

__global__ void
initializeMemory(float *dst, int size, float constant_val, float scaling_val)
{
  int tid = threadIdx.x;
  int tidx = blockDim.x * blockIdx.x + tid;
  if (tidx < size) {
    dst[tidx] = constant_val + scaling_val * tidx;
  }
}

void
initializeCudaMatrix(CudaMatrix *matrix, float constant_val, float scaling_val)
{
  initializeMemory<<<matrix->size / 64.0 + 1, 64>>>(matrix->ptr, matrix->size, constant_val, scaling_val);
}

void
printCudaMatrix(CudaMatrix *matrix)
{
  int n = matrix->dimension_sizes[0];
  int m = matrix->num_dimensions == 2 ? matrix->dimension_sizes[1] : 1;
  float *tmp = (float*) malloc(matrix->size * sizeof *tmp);
  CUDA_ERR_CHECK(hipMemcpy(tmp, matrix->ptr, matrix->size * sizeof *matrix->ptr, hipMemcpyDeviceToHost));
  printf("Printing CudaMatrix %p with sizes %dx%d\n",matrix, n, m);
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      printf("%.3f, ",tmp[i * n + j]);
    }
    printf("\n");
  }
  free(tmp);
}

void
addCudaElement(void *element, Destructor destructor_pointer)
{
  CudaQueueElement *new_element = (CudaQueueElement *) malloc(sizeof(CudaQueueElement));
  new_element->next = NULL;
  if (first == NULL) first = new_element;
  else {
    CudaQueueElement *current = first;
    while(current->next != NULL) {
      current = current->next;
    }
    current->next = new_element;
  }
  new_element->element = element;
  new_element->destructor_pointer = destructor_pointer;
}

void
destroyCudaMatrix(void *ptr)
{
  CudaMatrix *matrix = (CudaMatrix*) ptr;
  CUDA_ERR_CHECK(hipFree(matrix->ptr));
  free(matrix->dimension_sizes);
}

void
destroyCudaMemory(void *ptr)
{
  CUDA_ERR_CHECK(hipFree(ptr));
}

CudaMatrix *
createNdCudaMatrix(int n, ...)
{
  va_list args;
  va_start(args, n);
  int *dims = (int*) malloc(n * sizeof *dims);
  for (int i = 0; i < n; i++) {
    dims[i] = va_arg(args, int);
  }
  va_end(args);
  CudaMatrix *ret = createCudaMatrix(n, dims);
  return ret;
}

CudaMatrix *
create1dCudaMatrix(int a)
{
  int dims = 1, *dim = (int*) malloc(dims * sizeof *dim);
  dim[0] = a;
  CudaMatrix *ret = createCudaMatrix(dims, dim);
  return ret;
}

CudaMatrix *
create2dCudaMatrix(int a, int b)
{
  int dims = 2, *dim = (int*) malloc(dims * sizeof *dim);
  dim[0] = a;
  dim[1] = b;
  CudaMatrix *ret = createCudaMatrix(dims, dim);
  return ret;
}

CudaMatrix *
create3dCudaMatrix(int a, int b, int c)
{
  int dims = 3, *dim = (int*) malloc(dims * sizeof *dim);
  dim[0] = a;
  dim[1] = b;
  dim[2] = c;
  CudaMatrix *ret = createCudaMatrix(dims, dim);
  return ret;
}

CudaMatrix *
create4dCudaMatrix(int a, int b, int c, int d)
{
  int dims = 4, *dim = (int*) malloc(dims * sizeof *dim);
  dim[0] = a;
  dim[1] = b;
  dim[2] = c;
  dim[3] = d;
  CudaMatrix *ret = createCudaMatrix(dims, dim);
  return ret;
}

// This method steals the dimension_sizes pointer from caller.
CudaMatrix *
createCudaMatrix(int num_dimensions, int *dimension_sizes)
{
  CudaMatrix *ret;
  int n = 1;
  ret = (CudaMatrix *) malloc(sizeof *ret);
  ret->num_dimensions = num_dimensions;
  ret->dimension_sizes = dimension_sizes;
  for (int i = 0; i < num_dimensions; i++) {
    n *= dimension_sizes[i];
  }
  ret->size = n;
  CUDA_ERR_CHECK(hipMalloc(&ret->ptr, n * sizeof *ret->ptr));
  addCudaElement(ret, &destroyCudaMatrix);
  return ret;
}

void *
createCudaMemory(size_t size)
{
  void *ptr;
  CUDA_ERR_CHECK(hipMalloc(&ptr, size));
  addCudaElement(ptr, &destroyCudaMemory);
  return ptr;
}


void
destroyCudaPointers(void)
{
  CudaQueueElement *current = first, *next;
  while (current != NULL) {
    next = current->next;
    current->destructor_pointer(current->element);
    free(current);
    current = next;
  }
}
